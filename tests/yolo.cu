
#include <hip/hip_runtime.h>
__global__ void yolo(int *val) {
  for (int i = 0; i < 3; ++i) {
    if (threadIdx.x < 16)
      val[threadIdx.x + 1] = 0;
    else
      val[threadIdx.x] = 1;
    __syncthreads();
  }
}