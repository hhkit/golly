
#include <hip/hip_runtime.h>
__global__ void branch(int *val) {
  if (blockIdx.x > 2)
    if (threadIdx.x > 2)
      val[0] = val[1];
    else
      val[1] = 2;

  if (threadIdx.x == 0) {
    val[2] = 7;
  }

  if (threadIdx.x > 5) {
    for (int i = 0; i < 12; ++i) {
      val[i] = 18;
    }
  }

  for (int i = 0; i < 10; ++i) {
    if (i > 2)
      val[i] = 4;
    else
      val[i + 1] = 5;
  }
}